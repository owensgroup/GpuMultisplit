#include "hip/hip_runtime.h"
/*
GpuMultisplit is the proprietary property of The Regents of the University of California ("The Regents") and is copyright © 2016 The Regents of the University of California, Davis campus. All Rights Reserved. 

Redistribution and use in source and binary forms, with or without modification, are permitted by nonprofit educational or research institutions for noncommercial use only, provided that the following conditions are met:

* Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer. 
* Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other materials provided with the distribution. 
* The name or other trademarks of The Regents may not be used to endorse or promote products derived from this software without specific prior written permission.

The end-user understands that the program was developed for research purposes and is advised not to rely exclusively on the program for any reason.

THE SOFTWARE PROVIDED IS ON AN "AS IS" BASIS, AND THE REGENTS HAVE NO OBLIGATION TO PROVIDE MAINTENANCE, SUPPORT, UPDATES, ENHANCEMENTS, OR MODIFICATIONS. THE REGENTS SPECIFICALLY DISCLAIM ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE REGENTS BE LIABLE TO ANY PARTY FOR DIRECT, INDIRECT, SPECIAL, INCIDENTAL, EXEMPLARY OR CONSEQUENTIAL DAMAGES, INCLUDING BUT NOT LIMITED TO  PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES, LOSS OF USE, DATA OR PROFITS, OR BUSINESS INTERRUPTION, HOWEVER CAUSED AND UNDER ANY THEORY OF LIABILITY WHETHER IN CONTRACT, STRICT LIABILITY OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE AND ITS DOCUMENTATION, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

If you do not agree to these terms, do not download or use the software.  This license may be modified only in a writing signed by authorized signatory of both parties.

For license information please contact copyright@ucdavis.edu re T11-005.
*/

#include <stdio.h>
#include <iostream>
#include <stdint.h>
#include <cstring>
#include <cassert>
#include <hipcub/hipcub.hpp>
#include <functional>
#include "hip/hip_runtime_api.h"

#include "config/config_wms.h"
#include "api/wms_api.h"

#include "kernels/wms/wms_prescan.cuh"
#include "kernels/wms/wms_postscan.cuh"
#include "kernels/wms/wms_postscan_pairs.cuh"
#include "kernels/RBsort/reduced_bit_sort.cuh"
#include "cpu_functions.h"

// #define DEVICE_ID 0
// #define MY_ARCH_35__ // for K40c 
// // #define MY_ARCH_61__ // for GTX 1080

//===========================================================
template<typename key_type>
struct identity_bucket : public std::unary_function<key_type, uint32_t> {
   __forceinline__ __device__ __host__ uint32_t operator()(key_type a) const {
    return uint32_t(a);	
  }
};

struct delta_func : public std::unary_function<uint32_t, uint32_t> {
  delta_func(uint32_t delta) : delta_(delta) {}
  uint32_t delta_;
  __forceinline__ __device__ __host__ uint32_t operator()(uint32_t a) const {
    return (a/delta_);
  }
};

//===========================================================
// function declarations:
void random_input_generator(uint32_t* input, uint32_t n, uint32_t num_buckets, uint32_t log_buckets, uint32_t random_mode, uint32_t bucket_mode, uint32_t delta = 1, double alpha = 1.0);
//===========================================================
//===================
// main function:
//===================
int main(int argc, char** argv)
{
	int devCount;
  hipGetDeviceCount(&devCount);
  hipDeviceProp_t devProp;
  if(devCount){
    hipSetDevice(DEVICE_ID__); 
    hipGetDeviceProperties(&devProp, DEVICE_ID__);
  }
  printf("=====================================\n");
  printf("Device: %s\n", devProp.name);

  // ===============================
  srand(time(NULL));
  // srand(0);

	// number of input elements
	uint32_t n_elements = (1<<25);
	// number of buckets
	const uint32_t kNumBuckets = 2;
	const uint32_t kLogNumBuckets = int(ceil(log2(float(kNumBuckets))));
	uint32_t kIter = 1;
  if(cmdOptionExists(argv, argc+argv, "-iter"))
  	kIter = atoi(getCmdOption(argv, argv+argc, "-iter")); 

	uint32_t 	mode = 1;
  if(cmdOptionExists(argv, argc+argv, "-mode"))
    mode = atoi(getCmdOption(argv, argv+argc, "-mode"));

  // ==== simulation mode:
  // 1:		WMS key-only
  // 12 	WMS key-value
  // 2:		RBsort key-only
  // 22 	RBsort key-value

	printf("=====================================\n");
	printf("Mode %d \n", mode);
	printf("=====================================\n");
	switch(mode){
		case 1:
		printf("\t WMS: key-only\n");
		break;
		case 12:
		printf("\t WMS: key-value\n");
		break;
		case 2:
		printf("\t RBsort: key-only\n");
		break;
		case 22:
		printf("\t RBsort: key-value\n");
		break;
	}
	printf("=====================================\n");

	const bool 	is_protected = false; // if true, n_elements is cut to avoid non-complete subproblems (true was for dev mode, not being required currently)

	bool 			validate = true;
	bool 			debug_print = false;
  if(cmdOptionExists(argv, argc+argv, "-debug"))
	  debug_print = true;	

	// random input generator parameters:
	const uint32_t random_mode = 1;
	double alpha_hockey = 0.25;
	enum bucket_distribution{UNIFORM = 0, BINOMIAL = 1, HOCKEY = 2, UNIFORM_BUCKET = 3};

	// 1: random key generation within same width buckets (delta_bucket)
	uint32_t delta_buckets = (n_elements + kNumBuckets - 1)/kNumBuckets;
	// delta_func bucket_identifier(delta_buckets); // bucket identifier
	// bucket_distribution bucket_d = UNIFORM;

	// 2: Identity buckets
	identity_bucket<uint32_t> bucket_identifier;
	bucket_distribution bucket_d = UNIFORM_BUCKET;	

	printf("\t Number of buckets: %d\n", kNumBuckets);
	printf("\t Input distribution mode: %d\n", bucket_d);
	printf("\t UNIFORM = 0, BINOMIAL = 1, HOCKEY = 2, UNIFORM_BUCKET = 3\n");
	printf("=====================================\n");
	// === algorithm parameters: ================================

	uint32_t size_sub_prob = 1;
	uint32_t size_block = 1;
	if(mode == 1)
		size_sub_prob = subproblem_size_wms_key_only(kNumBuckets, size_block);
	else if (mode == 12)
		size_sub_prob = subproblem_size_wms_key_value(kNumBuckets, size_block);

	if(is_protected)
		n_elements = (n_elements/size_block)*size_block;
	if((mode != 2) && (mode != 22))
		assert((size_sub_prob != 1) && (size_block != 1));

	// =====================================================

	float temp_time = 0.0f;
	float pre_scan_time = 0.0f;
	float scan_time = 0.0f;
	float post_scan_time = 0.0f;

	float marking_reduced = 0.0f;
	float sorting_reduced = 0.0f;
	float packing_time = 0.0f;
	float unpacking_time = 0.0f;

	// ===============================
	// allocating memory:
	// ===============================
	uint32_t 	*h_key_in = new uint32_t[n_elements];
	uint32_t	*h_value_in = new uint32_t[n_elements];
	uint32_t	*h_value_out = new uint32_t[n_elements]; 	
	uint32_t 	*h_key_out = new uint32_t[n_elements];
	uint32_t 	*h_gpu_results_key = new uint32_t[n_elements];
	uint32_t 	*h_gpu_results_value = new uint32_t[n_elements];
	uint32_t	*h_cpu_results_key = NULL; // for validation
	uint32_t	*h_cpu_results_value = NULL; // for validation

	uint32_t* d_key_in;
	uint32_t* d_key_out;
	uint32_t* d_value_in;
	uint32_t* d_value_out;

	hipMalloc((void**)&d_key_in, sizeof(uint32_t) * n_elements);
	hipMalloc((void**)&d_key_out, sizeof(uint32_t) * n_elements);
	hipMalloc((void**)&d_value_in, sizeof(uint32_t) * n_elements);
	hipMalloc((void**)&d_value_out, sizeof(uint32_t) * n_elements);

	hipEvent_t start_pre, stop_pre, start_post, stop_post, start_scan, stop_scan;
	hipEventCreate(&start_pre);
	hipEventCreate(&start_post);
	hipEventCreate(&start_scan);
	hipEventCreate(&stop_pre);
	hipEventCreate(&stop_post);
	hipEventCreate(&stop_scan);

	if(mode == 1) // key-only Warp-wide Multisplit
	{
		uint32_t num_sub_prob_per_block = size_block/size_sub_prob;
		uint32_t num_sub_prob = (n_elements + size_sub_prob - 1)/(size_sub_prob);
		num_sub_prob = (num_sub_prob + num_sub_prob_per_block - 1)/num_sub_prob_per_block*num_sub_prob_per_block; // making sure block is full of subproblems, even if some will be invalidated afterwards.
		uint32_t num_blocks = (n_elements + size_block - 1)/size_block;
		printf("n = %d, num_blocks = %d, size_sub_prob = %d, num_sub_prob = %d\n", n_elements, num_blocks, size_sub_prob, num_sub_prob);		
		uint32_t* d_histogram;
		hipMalloc((void**)&d_histogram, sizeof(uint32_t)*kNumBuckets*num_sub_prob);

		void 		*d_temp_storage = NULL;
		size_t 	temp_storage_bytes = 0;

		hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_histogram, d_histogram, kNumBuckets * num_sub_prob);
		hipMalloc((void**)&d_temp_storage, temp_storage_bytes);

		if(validate)
			h_cpu_results_key = new uint32_t[n_elements];

		bool total_correctness = true;
		for(int kk = 0; kk<kIter; kk++)
		{
			random_input_generator(h_key_in, n_elements, kNumBuckets, kLogNumBuckets, bucket_d, random_mode, delta_buckets, alpha_hockey);
			hipMemcpy(d_key_in, h_key_in, sizeof(uint32_t) * n_elements, hipMemcpyHostToDevice);

			hipMemset(d_key_out, 0, sizeof(uint32_t)*n_elements);
			hipDeviceSynchronize();

			hipEventRecord(start_pre, 0);
			switch(kNumBuckets){
				case 2:
					if(is_protected)
						multisplit2_WMS_prescan<NUM_TILES_K_1, NUM_ROLLS_K_1, kNumBuckets , 1><<<num_blocks, 32*NUM_WARPS_K_1>>>(d_key_in, n_elements, d_histogram, bucket_identifier);
					else
						multisplit2_WMS_prescan_protected<NUM_TILES_K_1, NUM_ROLLS_K_1, kNumBuckets , 1><<<num_blocks, 32*NUM_WARPS_K_1>>>(d_key_in, n_elements, d_histogram, bucket_identifier);
				break;
				case 4:
					if(is_protected)
						multisplit2_WMS_prescan<NUM_TILES_K_2, NUM_ROLLS_K_2, kNumBuckets , 2><<<num_blocks, 32*NUM_WARPS_K_2>>>(d_key_in, n_elements, d_histogram, bucket_identifier);
					else
						multisplit2_WMS_prescan_protected<NUM_TILES_K_2, NUM_ROLLS_K_2, kNumBuckets , 2><<<num_blocks, 32*NUM_WARPS_K_2>>>(d_key_in, n_elements, d_histogram, bucket_identifier);												
				break;
				case 8:
					if(is_protected)
						multisplit2_WMS_prescan<NUM_TILES_K_3, NUM_ROLLS_K_3, kNumBuckets , 3><<<num_blocks, 32*NUM_WARPS_K_3>>>(d_key_in, n_elements, d_histogram, bucket_identifier);
					else
						multisplit2_WMS_prescan_protected<NUM_TILES_K_3, NUM_ROLLS_K_3, kNumBuckets , 3><<<num_blocks, 32*NUM_WARPS_K_3>>>(d_key_in, n_elements, d_histogram, bucket_identifier);	
				break;
				case 16:
					if(is_protected)
						multisplit2_WMS_prescan<NUM_TILES_K_4, NUM_ROLLS_K_4, kNumBuckets , 4><<<num_blocks, 32*NUM_WARPS_K_4>>>(d_key_in, n_elements, d_histogram, bucket_identifier);
					else
						multisplit2_WMS_prescan_protected<NUM_TILES_K_4, NUM_ROLLS_K_4, kNumBuckets , 4><<<num_blocks, 32*NUM_WARPS_K_4>>>(d_key_in, n_elements, d_histogram, bucket_identifier);	
				break;								
				case 32:
					if(is_protected)
						multisplit2_WMS_prescan<NUM_TILES_K_5, NUM_ROLLS_K_5, kNumBuckets , 5><<<num_blocks, 32*NUM_WARPS_K_5>>>(d_key_in, n_elements, d_histogram, bucket_identifier);
					else
						multisplit2_WMS_prescan_protected<NUM_TILES_K_5, NUM_ROLLS_K_5, kNumBuckets , 5><<<num_blocks, 32*NUM_WARPS_K_5>>>(d_key_in, n_elements, d_histogram, bucket_identifier);	
				break;				
			}
			hipEventRecord(stop_pre, 0);
			hipEventSynchronize(stop_pre);
			hipEventElapsedTime(&temp_time, start_pre, stop_pre);	
			pre_scan_time += temp_time;

			// printf("Histogram process finished in %.3f ms (%.3f Gkey/s)\n", pre_scan_time, float(n_elements)/pre_scan_time/1000.0f);

			if(debug_print){
				printf(" ### Input keys:\n");
				printGPUArray(d_key_in, n_elements, 32);
				hipMemset(d_key_out, 0, sizeof(uint32_t) * n_elements);
				printf(" ### GPU Histogram:\n");
				printGPUArray(d_histogram, num_sub_prob * kNumBuckets, 32);
			}
			hipEventRecord(start_scan, 0);
			hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_histogram, d_histogram, kNumBuckets * num_sub_prob);
			hipEventRecord(stop_scan, 0);
			hipEventSynchronize(stop_scan);
			hipEventElapsedTime(&temp_time, start_scan, stop_scan);	
			scan_time += temp_time;

			if(debug_print){
				printf("### GPU scanned histogram\n");
				printGPUArray(d_histogram, num_sub_prob * kNumBuckets, 32);
			}
			// post scan stage:
			hipEventRecord(start_post, 0);
			switch(kNumBuckets){
				case 2:
					if(is_protected)
						multisplit2_WMS_postscan_4rolls<NUM_WARPS_K_1, NUM_TILES_K_1, NUM_ROLLS_K_1, kNumBuckets, 1><<<num_blocks, 32*NUM_WARPS_K_1>>>(d_key_in, d_key_out, n_elements, d_histogram, bucket_identifier);
					else
						multisplit2_WMS_postscan_4rolls_protected<NUM_WARPS_K_1, NUM_TILES_K_1, NUM_ROLLS_K_1, kNumBuckets, 1><<<num_blocks, 32*NUM_WARPS_K_1>>>(d_key_in, d_key_out, n_elements, d_histogram, bucket_identifier);
				break;
				case 4:
					if(is_protected)
						multisplit2_WMS_postscan_4rolls<NUM_WARPS_K_2, NUM_TILES_K_2, NUM_ROLLS_K_2, kNumBuckets, 2><<<num_blocks, 32*NUM_WARPS_K_2>>>(d_key_in, d_key_out, n_elements, d_histogram, bucket_identifier);
					else
						multisplit2_WMS_postscan_4rolls_protected<NUM_WARPS_K_2, NUM_TILES_K_2, NUM_ROLLS_K_2, kNumBuckets, 2><<<num_blocks, 32*NUM_WARPS_K_2>>>(d_key_in, d_key_out, n_elements, d_histogram, bucket_identifier);												
				break;
				case 8:
					if(is_protected)
						multisplit2_WMS_postscan_4rolls<NUM_WARPS_K_3, NUM_TILES_K_3, NUM_ROLLS_K_3, kNumBuckets, 3><<<num_blocks, 32*NUM_WARPS_K_3>>>(d_key_in, d_key_out, n_elements, d_histogram, bucket_identifier);
					else
						multisplit2_WMS_postscan_4rolls_protected<NUM_WARPS_K_3, NUM_TILES_K_3, NUM_ROLLS_K_3, kNumBuckets, 3><<<num_blocks, 32*NUM_WARPS_K_3>>>(d_key_in, d_key_out, n_elements, d_histogram, bucket_identifier);												
				break;				
				case 16:
					if(is_protected)
						multisplit2_WMS_postscan_4rolls<NUM_WARPS_K_4, NUM_TILES_K_4, NUM_ROLLS_K_4, kNumBuckets, 4><<<num_blocks, 32*NUM_WARPS_K_4>>>(d_key_in, d_key_out, n_elements, d_histogram, bucket_identifier);
					else
						multisplit2_WMS_postscan_4rolls_protected<NUM_WARPS_K_4, NUM_TILES_K_4, NUM_ROLLS_K_4, kNumBuckets, 4><<<num_blocks, 32*NUM_WARPS_K_4>>>(d_key_in, d_key_out, n_elements, d_histogram, bucket_identifier);												
				break;				
				case 32:
					if(is_protected)
						multisplit2_WMS_postscan_4rolls<NUM_WARPS_K_5, NUM_TILES_K_5, NUM_ROLLS_K_5, kNumBuckets, 5><<<num_blocks, 32*NUM_WARPS_K_5>>>(d_key_in, d_key_out, n_elements, d_histogram, bucket_identifier);
					else
						multisplit2_WMS_postscan_4rolls_protected<NUM_WARPS_K_5, NUM_TILES_K_5, NUM_ROLLS_K_5, kNumBuckets, 5><<<num_blocks, 32*NUM_WARPS_K_5>>>(d_key_in, d_key_out, n_elements, d_histogram, bucket_identifier);												
				break;				
			}
			hipEventRecord(stop_post, 0);
			hipEventSynchronize(stop_post);
			hipEventElapsedTime(&temp_time, start_post, stop_post);	
			post_scan_time += temp_time;	
			
			if(debug_print){
				printf(" ### Output keys:\n");
				printGPUArray(d_key_out, n_elements, 32);
			}
			if(validate)
			{
				cpu_multisplit_general(h_key_in, h_cpu_results_key, n_elements, bucket_identifier, 0, kNumBuckets);
				hipMemcpy(h_gpu_results_key, d_key_out, sizeof(uint32_t) * n_elements, hipMemcpyDeviceToHost);
				bool correct = true;
				for(int i = 0; i<n_elements && correct;i++)
				{
					if(h_cpu_results_key[i] != h_gpu_results_key[i]){
						printf(" ### Iteration %d: Wrong results at index %d: cpu = %d, gpu = %d\n", kk, i, h_cpu_results_key[i], h_gpu_results_key[i]);
						correct = false;
					}
				}
				total_correctness &= correct;
			}
		}
		pre_scan_time /= kIter;	
		scan_time /= kIter;
		post_scan_time /= kIter;

		float total_time = pre_scan_time + post_scan_time + scan_time;
		printf("WMS key-only with %d buckets finished in %.3f ms, and %.3f Mkey/s\n", kNumBuckets, total_time, float(n_elements)/total_time/1000.0f);
		printf("\t Pre scan %.3f ms (%.2f)\n", pre_scan_time, float(pre_scan_time)/float(total_time));
		printf("\t Scan %.3f ms (%.2f)\n", scan_time, float(scan_time)/float(total_time));
		printf("\t Post scan %.3f ms (%.2f)\n", post_scan_time, float(post_scan_time)/float(total_time));

		if(validate)
		{
			if(total_correctness) printf("Validation was done successfully!\n");
			else printf("Validation failed!\n");			
		}

		//====================================
		//==
		hipFree(d_histogram);
		hipFree(d_temp_storage);			
	}
	else if(mode == 12)
	{
		uint32_t num_sub_prob_per_block = size_block/size_sub_prob;
		uint32_t num_sub_prob = (n_elements + size_sub_prob - 1)/(size_sub_prob);
		num_sub_prob = (num_sub_prob + num_sub_prob_per_block - 1)/num_sub_prob_per_block*num_sub_prob_per_block; // making sure block is full of subproblems, even if some will be invalidated afterwards.
		uint32_t num_blocks = (n_elements + size_block - 1)/size_block;
		printf("n = %d, num_blocks = %d, size_sub_prob = %d, num_sub_prob = %d\n", n_elements, num_blocks, size_sub_prob, num_sub_prob);		
		uint32_t* d_histogram;
		hipMalloc((void**)&d_histogram, sizeof(uint32_t)*kNumBuckets*num_sub_prob);

		void 		*d_temp_storage = NULL;
		size_t 	temp_storage_bytes = 0;

		hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_histogram, d_histogram, kNumBuckets * num_sub_prob);
		hipMalloc((void**)&d_temp_storage, temp_storage_bytes);

		if(validate)
		{
			h_cpu_results_key = new uint32_t[n_elements];
			h_cpu_results_value = new uint32_t[n_elements];			
		}			

		bool total_correctness = true;
		for(int kk = 0; kk<kIter; kk++)
		{
			random_input_generator(h_key_in, n_elements, kNumBuckets, kLogNumBuckets, bucket_d, random_mode, delta_buckets, alpha_hockey);
			for(int k = 0; k<n_elements;k++)
				h_value_in[k] = h_key_in[k];
			hipMemcpy(d_key_in, h_key_in, sizeof(uint32_t) * n_elements, hipMemcpyHostToDevice);
			hipMemcpy(d_value_in, h_value_in, sizeof(uint32_t) * n_elements, hipMemcpyHostToDevice);
			hipMemset(d_key_out, 0, sizeof(uint32_t)*n_elements);
			hipMemset(d_value_out, 0, sizeof(uint32_t)*n_elements);
			hipDeviceSynchronize();

			hipEventRecord(start_pre, 0);
			switch(kNumBuckets){
				case 2:
				if(is_protected)
					multisplit2_WMS_prescan<NUM_TILES_KV_1, NUM_ROLLS_KV_1, kNumBuckets , 1><<<num_blocks, 32*NUM_WARPS_KV_1>>>(d_key_in, n_elements, d_histogram, bucket_identifier);
				else
					multisplit2_WMS_prescan_protected<NUM_TILES_KV_1, NUM_ROLLS_KV_1, kNumBuckets , 1><<<num_blocks, 32*NUM_WARPS_KV_1>>>(d_key_in, n_elements, d_histogram, bucket_identifier);					
				break;
				case 4:
				if(is_protected)
					multisplit2_WMS_prescan<NUM_TILES_KV_2, NUM_ROLLS_KV_2, kNumBuckets , 2><<<num_blocks, 32*NUM_WARPS_KV_2>>>(d_key_in, n_elements, d_histogram, bucket_identifier);
				else
					multisplit2_WMS_prescan_protected<NUM_TILES_KV_2, NUM_ROLLS_KV_2, kNumBuckets , 2><<<num_blocks, 32*NUM_WARPS_KV_2>>>(d_key_in, n_elements, d_histogram, bucket_identifier);					
				break;
				case 8:
				if(is_protected)
					multisplit2_WMS_prescan<NUM_TILES_KV_3, NUM_ROLLS_KV_3, kNumBuckets , 3><<<num_blocks, 32*NUM_WARPS_KV_3>>>(d_key_in, n_elements, d_histogram, bucket_identifier);
				else
					multisplit2_WMS_prescan_protected<NUM_TILES_KV_3, NUM_ROLLS_KV_3, kNumBuckets , 3><<<num_blocks, 32*NUM_WARPS_KV_3>>>(d_key_in, n_elements, d_histogram, bucket_identifier);					
				break;
				case 16:
				if(is_protected)
					multisplit2_WMS_prescan<NUM_TILES_KV_4, NUM_ROLLS_KV_4, kNumBuckets , 4><<<num_blocks, 32*NUM_WARPS_KV_4>>>(d_key_in, n_elements, d_histogram, bucket_identifier);
				else
					multisplit2_WMS_prescan_protected<NUM_TILES_KV_4, NUM_ROLLS_KV_4, kNumBuckets , 4><<<num_blocks, 32*NUM_WARPS_KV_4>>>(d_key_in, n_elements, d_histogram, bucket_identifier);					
				break;
				case 32:
				if(is_protected)
					multisplit2_WMS_prescan<NUM_TILES_KV_5, NUM_ROLLS_KV_5, kNumBuckets , 5><<<num_blocks, 32*NUM_WARPS_KV_5>>>(d_key_in, n_elements, d_histogram, bucket_identifier);
				else
					multisplit2_WMS_prescan_protected<NUM_TILES_KV_5, NUM_ROLLS_KV_5, kNumBuckets , 5><<<num_blocks, 32*NUM_WARPS_KV_5>>>(d_key_in, n_elements, d_histogram, bucket_identifier);					
				break;																
			}
			hipEventRecord(stop_pre, 0);
			hipEventSynchronize(stop_pre);
			hipEventElapsedTime(&temp_time, start_pre, stop_pre);	
			pre_scan_time += temp_time;

			// printf("Histogram process finished in %.3f ms (%.3f Gkey/s)\n", pre_scan_time, float(n_elements)/pre_scan_time/1000.0f);

			if(debug_print){
				printf(" ### Input keys:\n");
				printGPUArray(d_key_in, n_elements, 32);
				hipMemset(d_key_out, 0, sizeof(uint32_t) * n_elements);
				printf(" ### GPU Histogram:\n");
				printGPUArray(d_histogram, num_sub_prob * kNumBuckets, 32);
			}
			hipEventRecord(start_scan, 0);
			hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_histogram, d_histogram, kNumBuckets * num_sub_prob);
			hipEventRecord(stop_scan, 0);
			hipEventSynchronize(stop_scan);
			hipEventElapsedTime(&temp_time, start_scan, stop_scan);	
			scan_time += temp_time;

			if(debug_print){
				printf("### GPU scanned histogram\n");
				printGPUArray(d_histogram, num_sub_prob * kNumBuckets, 32);
			}
			// post scan stage:
			hipEventRecord(start_post, 0);
			switch(kNumBuckets){
				case 2:
				if(is_protected)
						multisplit2_WMS_postscan_4rolls_pairs<NUM_WARPS_KV_1, NUM_TILES_KV_1, NUM_ROLLS_KV_1, kNumBuckets, 1><<<num_blocks, 32*NUM_WARPS_KV_1>>>(d_key_in, d_value_in, d_key_out, d_value_out, n_elements, d_histogram, bucket_identifier);				
					else
						multisplit2_WMS_postscan_4rolls_pairs_protected<NUM_WARPS_KV_1, NUM_TILES_KV_1, NUM_ROLLS_KV_1, kNumBuckets, 1><<<num_blocks, 32*NUM_WARPS_KV_1>>>(d_key_in, d_value_in, d_key_out, d_value_out, n_elements, d_histogram, bucket_identifier);						
				break;
				case 4:
				if(is_protected)
						multisplit2_WMS_postscan_4rolls_pairs<NUM_WARPS_KV_2, NUM_TILES_KV_2, NUM_ROLLS_KV_2, kNumBuckets, 2><<<num_blocks, 32*NUM_WARPS_KV_2>>>(d_key_in, d_value_in, d_key_out, d_value_out, n_elements, d_histogram, bucket_identifier);				
					else
						multisplit2_WMS_postscan_4rolls_pairs_protected<NUM_WARPS_KV_2, NUM_TILES_KV_2, NUM_ROLLS_KV_2, kNumBuckets, 2><<<num_blocks, 32*NUM_WARPS_KV_2>>>(d_key_in, d_value_in, d_key_out, d_value_out, n_elements, d_histogram, bucket_identifier);						
				break;
				case 8:
				if(is_protected)
						multisplit2_WMS_postscan_4rolls_pairs<NUM_WARPS_KV_3, NUM_TILES_KV_3, NUM_ROLLS_KV_3, kNumBuckets, 3><<<num_blocks, 32*NUM_WARPS_KV_3>>>(d_key_in, d_value_in, d_key_out, d_value_out, n_elements, d_histogram, bucket_identifier);				
					else
						multisplit2_WMS_postscan_4rolls_pairs_protected<NUM_WARPS_KV_3, NUM_TILES_KV_3, NUM_ROLLS_KV_3, kNumBuckets, 3><<<num_blocks, 32*NUM_WARPS_KV_3>>>(d_key_in, d_value_in, d_key_out, d_value_out, n_elements, d_histogram, bucket_identifier);						
				break;							
				case 16:
				if(is_protected)
						multisplit2_WMS_postscan_4rolls_pairs<NUM_WARPS_KV_4, NUM_TILES_KV_4, NUM_ROLLS_KV_4, kNumBuckets, 4><<<num_blocks, 32*NUM_WARPS_KV_4>>>(d_key_in, d_value_in, d_key_out, d_value_out, n_elements, d_histogram, bucket_identifier);				
					else
						multisplit2_WMS_postscan_4rolls_pairs_protected<NUM_WARPS_KV_4, NUM_TILES_KV_4, NUM_ROLLS_KV_4, kNumBuckets, 4><<<num_blocks, 32*NUM_WARPS_KV_4>>>(d_key_in, d_value_in, d_key_out, d_value_out, n_elements, d_histogram, bucket_identifier);						
				break;					
				case 32:
				if(is_protected)
						multisplit2_WMS_postscan_4rolls_pairs<NUM_WARPS_KV_5, NUM_TILES_KV_5, NUM_ROLLS_KV_5, kNumBuckets, 5><<<num_blocks, 32*NUM_WARPS_KV_5>>>(d_key_in, d_value_in, d_key_out, d_value_out, n_elements, d_histogram, bucket_identifier);				
					else
						multisplit2_WMS_postscan_4rolls_pairs_protected<NUM_WARPS_KV_5, NUM_TILES_KV_5, NUM_ROLLS_KV_5, kNumBuckets, 5><<<num_blocks, 32*NUM_WARPS_KV_5>>>(d_key_in, d_value_in, d_key_out, d_value_out, n_elements, d_histogram, bucket_identifier);						
				break;				
			}
			hipEventRecord(stop_post, 0);
			hipEventSynchronize(stop_post);
			hipEventElapsedTime(&temp_time, start_post, stop_post);	
			post_scan_time += temp_time;	
			
			if(debug_print){
				printf(" ### Output keys:\n");
				printGPUArray(d_key_out, n_elements, 32);
			}
			if(validate)
			{
				cpu_multisplit_pairs_general(h_key_in, h_cpu_results_key, h_value_in, h_cpu_results_value, n_elements, bucket_identifier, 0, kNumBuckets);
				hipMemcpy(h_gpu_results_key, d_key_out, sizeof(uint32_t) * n_elements, hipMemcpyDeviceToHost);
				hipMemcpy(h_gpu_results_value, d_value_out, sizeof(uint32_t) * n_elements, hipMemcpyDeviceToHost);
				bool correct = true;
				for(int i = 0; i<n_elements && correct;i++)
				{
					if((h_cpu_results_key[i] != h_gpu_results_key[i]) || (h_cpu_results_value[i] != h_gpu_results_value[i])){
						printf(" ### Wrong results at index %d: cpu = (%d, %d), gpu = (%d,%d)\n", i, h_cpu_results_key[i], h_cpu_results_value[i], h_gpu_results_key[i], h_gpu_results_value[i]);
						correct = false;
					}				
				}
				total_correctness &= correct;
			}
		}
		pre_scan_time /= kIter;	
		scan_time /= kIter;
		post_scan_time /= kIter;

		float total_time = pre_scan_time + post_scan_time + scan_time;
		printf("WMS key-value with %d buckets finished in %.3f ms, and %.3f Mkey/s\n", kNumBuckets, total_time, float(n_elements)/total_time/1000.0f);
		printf("\t Pre scan %.3f ms (%.2f)\n", pre_scan_time, float(pre_scan_time)/float(total_time));
		printf("\t Scan %.3f ms (%.2f)\n", scan_time, float(scan_time)/float(total_time));
		printf("\t Post scan %.3f ms (%.2f)\n", post_scan_time, float(post_scan_time)/float(total_time));

		if(validate)
		{
			if(total_correctness) printf("Validation was done successfully!\n");
			else printf("Validation failed!\n");			
		}
		//==============================
		hipFree(d_histogram);
		hipFree(d_temp_storage);					
	}
	else if(mode == 2) // reduced-bit sort (key-only)
	{
		void 		*d_temp_storage = NULL;
		size_t 	temp_storage_bytes = 0;
		uint32_t* d_bucket_in;
		uint32_t* d_bucket_out;
		hipMalloc((void**)&d_bucket_in, sizeof(uint32_t) * n_elements);
		hipMalloc((void**)&d_bucket_out, sizeof(uint32_t) * n_elements);
		hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_bucket_in, d_bucket_out, d_key_in, d_key_out, n_elements, 0, kLogNumBuckets);
		hipMalloc((void**)&d_temp_storage, temp_storage_bytes);

		if(validate)
		{
			h_cpu_results_key = new uint32_t[n_elements];
		}	

		bool total_correctness = true;
		uint32_t num_blocks = (n_elements + NUM_THREADS_REDUCED - 1)/NUM_THREADS_REDUCED;
		for(int kk = 0; kk<kIter; kk++)
		{
			hipMemset(d_key_out, 0, sizeof(uint32_t)*n_elements);

			// generating keys:
			random_input_generator(h_key_in, n_elements, kNumBuckets, kLogNumBuckets, bucket_d, random_mode, delta_buckets, alpha_hockey);
			hipMemcpy(d_key_in, h_key_in, sizeof(uint32_t) * n_elements, hipMemcpyHostToDevice);
			hipDeviceSynchronize();
			if(debug_print){
				printf(" ### Input keys:\n");
				printGPUArray(d_key_in, n_elements, 32);
			}

			// Reduced-bit sort method (key-only)
			// == marking buckets:
		  hipEventRecord(start_pre, 0);
			markBins_general<<<num_blocks, NUM_THREADS_REDUCED>>>(d_bucket_in, d_key_in, n_elements, kNumBuckets, bucket_identifier);
			hipEventRecord(stop_pre,0);
			hipEventSynchronize(stop_pre);
			hipEventElapsedTime(&temp_time, start_pre, stop_pre);
			marking_reduced += temp_time;

			if(debug_print)
			{
				printf(" ### Buckets:\n");
				printGPUArray(d_bucket_in, n_elements, 32);				
			}
			// == sorting bucketIds
			hipEventRecord(start_post, 0);
			hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_bucket_in, d_bucket_out, d_key_in, d_key_out, n_elements, 0, kLogNumBuckets);
			hipEventRecord(stop_post,0);
			hipEventSynchronize(stop_post);
			hipEventElapsedTime(&temp_time, start_post, stop_post);
			sorting_reduced += temp_time;

			if(debug_print)
			{
				printf(" ### Output keys:\n");
				printGPUArray(d_key_out, n_elements, 32);								
			}
			if(validate)
			{
				cpu_multisplit_general(h_key_in, h_cpu_results_key, n_elements, bucket_identifier, 0, kNumBuckets);
				hipMemcpy(h_gpu_results_key, d_key_out, sizeof(uint32_t) * n_elements, hipMemcpyDeviceToHost);
				bool correct = true;
				for(int i = 0; i<n_elements && correct;i++)
				{
					if(h_cpu_results_key[i] != h_gpu_results_key[i]){
						printf(" ### Iteration %d: Wrong results at index %d: cpu = %d, gpu = %d\n", kk, i, h_cpu_results_key[i], h_gpu_results_key[i]);
						correct = false;
					}
				}
				total_correctness &= correct;
			}
		}
		marking_reduced /= kIter;
		sorting_reduced /= kIter;
		float reduced_total_time = marking_reduced + sorting_reduced;

		printf("Reduced-bit sort key-only with %d buckets finished in %.3f ms, and %.3f Mkey/s\n", kNumBuckets, reduced_total_time, float(n_elements)/reduced_total_time/1000.0f);
		printf("\t Marking %.3f ms (%.2f)\n", marking_reduced, float(marking_reduced)/float(reduced_total_time));
		printf("\t Sorting %.3f ms (%.2f)\n", sorting_reduced, float(sorting_reduced)/float(reduced_total_time));

		if(validate)
		{
			if(total_correctness) printf("Validation was done successfully!\n");
			else printf("Validation failed!\n");			
		}

		if(d_bucket_in) hipFree(d_bucket_in);
		if(d_bucket_out) hipFree(d_bucket_out);
		if(d_temp_storage) hipFree(d_temp_storage);
	}
	else if(mode == 22) // reduced-bit sort (key-value)
	{
		void 		*d_temp_storage = NULL;
		size_t 	temp_storage_bytes = 0;
		uint32_t* d_bucket_in;
		uint32_t* d_bucket_out;
		uint64_t* d_temp_in;
		uint64_t* d_temp_out;

		hipMalloc((void**)&d_bucket_in, sizeof(uint32_t) * n_elements);
		hipMalloc((void**)&d_bucket_out, sizeof(uint32_t) * n_elements);
		hipMalloc((void**)&d_temp_in, sizeof(uint64_t) * n_elements);
		hipMalloc((void**)&d_temp_out, sizeof(uint64_t) * n_elements);

		hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_bucket_in, d_bucket_out, d_temp_in, d_temp_out, n_elements, 0, kLogNumBuckets);
		hipMalloc((void**)&d_temp_storage, temp_storage_bytes);

		if(validate)
		{
			h_cpu_results_key = new uint32_t[n_elements];
			h_cpu_results_value = new uint32_t[n_elements];
		}	

		bool total_correctness = true;
		uint32_t num_blocks = (n_elements + NUM_THREADS_REDUCED - 1)/NUM_THREADS_REDUCED;
		for(int kk = 0; kk<kIter; kk++)
		{
			hipMemset(d_key_out, 0, sizeof(uint32_t)*n_elements);
			hipMemset(d_value_out, 0, sizeof(uint32_t)*n_elements);

			// generating keys:
			random_input_generator(h_key_in, n_elements, kNumBuckets, kLogNumBuckets, bucket_d, random_mode, delta_buckets, alpha_hockey);
			for(int i = 0; i<n_elements; i++)
				h_value_in[i] = h_key_in[i];

			hipMemcpy(d_key_in, h_key_in, sizeof(uint32_t) * n_elements, hipMemcpyHostToDevice);
			hipMemcpy(d_value_in, h_value_in, sizeof(uint32_t) * n_elements, hipMemcpyHostToDevice);
			hipDeviceSynchronize();

			if(debug_print){
				printf(" ### Input keys:\n");
				printGPUArray(d_key_in, n_elements, 32);
				printf(" ### Input values:\n");
				printGPUArray(d_value_in, n_elements, 32);				
			}

			// Reduced-bit sort method (key-only)
			// == marking buckets:
		  hipEventRecord(start_pre, 0);
			markBins_general<<<num_blocks, NUM_THREADS_REDUCED>>>(d_bucket_in, d_key_in, n_elements, kNumBuckets, bucket_identifier);
			hipEventRecord(stop_pre,0);
			hipEventSynchronize(stop_pre);
			hipEventElapsedTime(&temp_time, start_pre, stop_pre);
			marking_reduced += temp_time;

			if(debug_print)
			{
				printf(" ### Buckets:\n");
				printGPUArray(d_bucket_in, n_elements, 32);				
			}

			hipEventRecord(start_pre, 0);
			packingKeyValuePairs<<<num_blocks, NUM_THREADS_REDUCED>>>(d_temp_in, d_key_in, d_value_in,n_elements);
			hipEventRecord(stop_pre,0);
			hipEventSynchronize(stop_pre);
			hipEventElapsedTime(&temp_time, start_pre, stop_pre);
			packing_time += temp_time;
			
			// == sorting bucketIds
			hipEventRecord(start_post, 0);
			hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_bucket_in, d_bucket_out, d_temp_in, d_temp_out, n_elements, 0, kLogNumBuckets);
			hipEventRecord(stop_post,0);
			hipEventSynchronize(stop_post);
			hipEventElapsedTime(&temp_time, start_post, stop_post);
			sorting_reduced += temp_time;

			hipEventRecord(start_post, 0);
			unpackingKeyValuePairs<<<num_blocks, NUM_THREADS_REDUCED>>>(d_temp_out, d_key_out, d_value_out, n_elements);
			hipEventRecord(stop_post,0);
			hipEventSynchronize(stop_post);
			hipEventElapsedTime(&temp_time, start_post, stop_post);
			unpacking_time += temp_time;

			if(debug_print)
			{
				printf(" ### Output keys:\n");
				printGPUArray(d_key_out, n_elements, 32);								
				printf(" ### Output value:\n");
				printGPUArray(d_value_out, n_elements, 32);								
			}

			if(validate)
			{
				cpu_multisplit_pairs_general(h_key_in, h_cpu_results_key, h_value_in, h_cpu_results_value, n_elements, bucket_identifier, 0, kNumBuckets);
				hipMemcpy(h_gpu_results_key, d_key_out, sizeof(uint32_t) * n_elements, hipMemcpyDeviceToHost);
				hipMemcpy(h_gpu_results_value, d_value_out, sizeof(uint32_t) * n_elements, hipMemcpyDeviceToHost);
				bool correct = true;
				for(int i = 0; i<n_elements && correct;i++)
				{
					if((h_cpu_results_key[i] != h_gpu_results_key[i]) || (h_cpu_results_value[i] != h_gpu_results_value[i])){
						printf(" ### Wrong results at index %d: cpu = (%d, %d), gpu = (%d,%d)\n", i, h_cpu_results_key[i], h_cpu_results_value[i], h_gpu_results_key[i], h_gpu_results_value[i]);
						correct = false;
					}				
				}
				total_correctness &= correct;
			}
		}
		marking_reduced /= kIter;
		sorting_reduced /= kIter;
		packing_time	/= kIter;
		unpacking_time /= kIter;

		float reduced_total_time = marking_reduced + sorting_reduced + packing_time + unpacking_time;

		printf("Reduced-bit sort key-value with %d buckets finished in %.3f ms, and %.3f Mkey/s\n", kNumBuckets, reduced_total_time, float(n_elements)/reduced_total_time/1000.0f);
		printf("\t Marking %.3f ms (%.2f)\n", marking_reduced, float(marking_reduced)/float(reduced_total_time));
		printf("\t Packing/unpacking %.3f / %.3f ms (%.3f ms total): %.2f\n", packing_time, unpacking_time, packing_time + unpacking_time, float(packing_time + unpacking_time)/float(reduced_total_time));
		printf("\t Sorting %.3f ms (%.2f)\n", sorting_reduced, float(sorting_reduced)/float(reduced_total_time));

		if(validate)
		{
			if(total_correctness) printf("Validation was done successfully!\n");
			else printf("Validation failed!\n");			
		}

		if(d_bucket_in) hipFree(d_bucket_in);
		if(d_bucket_out) hipFree(d_bucket_out);
		if(d_temp_in) hipFree(d_temp_in);
		if(d_temp_out) hipFree(d_temp_out);
		if(d_temp_storage) hipFree(d_temp_storage);
	}	
	// ===============================
	// releasing memory:
	// ===============================
	hipEventDestroy(start_pre);
	hipEventDestroy(start_scan);
	hipEventDestroy(start_post);
	hipEventDestroy(stop_pre);
	hipEventDestroy(stop_scan);
	hipEventDestroy(stop_post);

	if(h_key_in) delete[] h_key_in;
	if(h_key_out) delete[] h_key_out;
	if(h_value_in) delete[] h_value_in;
	if(h_value_out) delete[] h_value_out;

	if(h_gpu_results_key) delete[] h_gpu_results_key;
	if(h_cpu_results_key) delete[] h_cpu_results_key;
	if(h_gpu_results_value) delete[] h_gpu_results_value;
	if(h_cpu_results_value) delete[] h_cpu_results_value;

	if(d_key_in) hipFree(d_key_in);
	if(d_key_out) hipFree(d_key_out);
	if(d_value_in) hipFree(d_value_in);
	if(d_value_out) hipFree(d_value_out);
}
